#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <map>
#include <climits> 
#include <cfloat>
#include <vector>
#include <algorithm> // for heap
#include <numeric> // std::iota
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include "libarff/arff_parser.h"
#include "libarff/arff_data.h"

#define DISTANCETHREADSPERBLOCKDIM 16  // TODO change from 32 to 16? why isnt distance 32?
#define REDUCTIONTHREADSBLOCKDIMX 256
#define REDUCTIONTHREADSBLOCKDIMY 4

__global__ void fillDistanceMatrix(float *d_datasetArray, float *d_distanceMatrix, int width, int numberOfAttributes) { // d_distanceMatrix is a square matrix
	int column = ( blockDim.x * blockIdx.x ) + threadIdx.x; // TODO which is outer and inner?
	int row    = ( blockDim.y * blockIdx.y ) + threadIdx.y;

	if (row < width && column < width) {
		if (row == column) {
			d_distanceMatrix[row*width + column] = FLT_MAX; // cant compare to to self
		} else {
			float distance = 0;

			for(int k = 0; k < numberOfAttributes - 1; k++) { // compute the distance between the two instances
				float diff = d_datasetArray[row * numberOfAttributes + k] - d_datasetArray[column * numberOfAttributes + k]; // one instance minus the other
				distance += diff * diff;
			}

			d_distanceMatrix[row*width + column] = sqrt(distance);
		}
	}
}

// Uses a shared memory reduction approach to find the smallest k values
__global__ void deviceFindMinK(float *d_smallestK, int *d_smallestKClasses, float *d_distanceMatrix, int *d_actualClasses, int numInstances, int k) {
	__shared__ float sharedDistanceMemory[REDUCTIONTHREADSBLOCKDIMY][REDUCTIONTHREADSBLOCKDIMX];
	__shared__ int sharedClassMemory[REDUCTIONTHREADSBLOCKDIMY][REDUCTIONTHREADSBLOCKDIMX];

	int tid_x = blockIdx.x*blockDim.x + threadIdx.x;
	int tid_y  = blockIdx.y*blockDim.y + threadIdx.y;
	int startingDistanceIndex = tid_y * numInstances; // NOTE startingDistanceIndex is used to access the proper "row" of the distance matrix

	if (tid_x == 0 && tid_y == 0) {
		printf("\nin deviceFindMinK blockDim.x: %d, blockDim.y: %d, gridDim.x: %d, gridDim.y: %d:\n", blockDim.x, blockDim.y, gridDim.x, gridDim.y);
	}

	sharedDistanceMemory[threadIdx.y][threadIdx.x] = (tid_x < numInstances) && (tid_y < numInstances) ? d_distanceMatrix[startingDistanceIndex + tid_x] : FLT_MAX;
	sharedClassMemory[threadIdx.y][threadIdx.x] = (tid_x < numInstances) && (tid_y < numInstances) ? d_actualClasses[tid_x] : -1;

	__syncthreads();

	// do reduction in shared memory
	// for (int s = 0; s < blockDim.x; s += k) {
	// ((ceiling division of blockDim.x / k) / 2) is number of "chunks" of size k that barely spill over the halfway point
	// mulitply it by k to get the actual max s value to start at
	int prevS = blockDim.x; // TODO works at max k?

	for (int s = (((blockDim.x + k - 1) / k) / 2) * k; s < prevS; s = (((s / k) + 2 - 1) / 2) * k) { // (ceil(blocksSizeK left / 2) * k)  TODO what happens when k > blockDim?
		if (tid_x == 0 && tid_y == 0) {
			printf("\nhere1\n");
		}
		if (threadIdx.x < s && threadIdx.x % k == 0 && tid_y < numInstances) { // TODO check if tid_y < numInstances? to prevent precitions for the excess 'y' dim blocks?
			int leftIndex = threadIdx.x;
			int rightIndex = leftIndex + s;
			// printf("s: %d, leftIndex: %d, rightIndex: %d\n", s, leftIndex, rightIndex);
			float* result = new float[k]; // TODO does something need to be freed?
			int* resultClasses = new int[k]; // TODO does something need to be freed?

			// if on first iteration
			if (prevS == blockDim.x) {
				thrust::sort_by_key(thrust::seq, sharedDistanceMemory[threadIdx.y] + leftIndex, sharedDistanceMemory[threadIdx.y] + leftIndex + k, sharedClassMemory[threadIdx.y] + leftIndex);
				int actualEndingIndex = rightIndex + k;
				if (actualEndingIndex >= REDUCTIONTHREADSBLOCKDIMX)
					actualEndingIndex = REDUCTIONTHREADSBLOCKDIMX;
				thrust::sort_by_key(thrust::seq, sharedDistanceMemory[threadIdx.y] + rightIndex, sharedDistanceMemory[threadIdx.y] + actualEndingIndex, sharedClassMemory[threadIdx.y] + rightIndex);
			}

			if (tid_x == 0 && tid_y == 0) {
				printf("\nhere2\n");
			}

			for (int i = 0; i < k; i++) {
				if (rightIndex < blockDim.x && sharedDistanceMemory[threadIdx.y][rightIndex] < sharedDistanceMemory[threadIdx.y][leftIndex]) {
					result[i] = sharedDistanceMemory[threadIdx.y][rightIndex];
					resultClasses[i] = sharedClassMemory[threadIdx.y][rightIndex];
					rightIndex++;
				} else {
					result[i] = sharedDistanceMemory[threadIdx.y][leftIndex];
					resultClasses[i] = sharedClassMemory[threadIdx.y][leftIndex];
					leftIndex++;
				}
			}

			// if (tid_x == 0 && tid_y == 0) {
			// 	printf("\nhere4\n");
			// }

			for (int i = 0; i < k; i++) {
				sharedDistanceMemory[threadIdx.y][threadIdx.x + i] = result[i];
				sharedClassMemory[threadIdx.y][threadIdx.x + i] = resultClasses[i];
			}
		}
		
		prevS = s;

		__syncthreads();
	}

	// if (tid_x == 256 && tid_y == 0) {
	// 	int start = 0;
	// 	int end = start + k;
	// 	printf("\n block final smallest k:\n");
	// 	for (int i = start; i < end; i++) {
	// 		printf("%f, ", sharedDistanceMemory[threadIdx.y][i]);
	// 	}
	// 	printf("\n");
	// 	printf("\n block final classes of smallest k:\n");
	// 	for (int i = start; i < end; i++) {
	// 		printf("%d, ", sharedClassMemory[threadIdx.y][i]);
	// 	}
	// 	printf("\n");
	// }

	// write your nearestK to global mem
	if (threadIdx.x == 0 && tid_y < numInstances) {
		int startingKIndex = ((tid_y * gridDim.x) + (tid_x/blockDim.x)) * k;
		int endingKIndex = startingKIndex + k;
		// printf("tid_y: %d, startingKIndex: %d\n", tid_y, startingKIndex);

		int j = 0;
		for (int i = startingKIndex; i < endingKIndex; i++) {
			d_smallestK[i] = sharedDistanceMemory[threadIdx.y][j];
			d_smallestKClasses[i] = sharedClassMemory[threadIdx.y][j];
			j++;
		}
	}

	// __syncthreads();

	// if (tid_x == 0 && tid_y == 335) {
	// 	printf("\n d_smallestK final smallest k:\n");
	// 	for (int i = 0; i < 4 * k; i++) {
	// 		printf("%f, ", d_smallestK[i]);
	// 	}
	// 	printf("\n");
	// 	printf("\n d_smallestKClasses final smallest k:\n");
	// 	for (int i = 0; i < 4 * k; i++) {
	// 		printf("%d, ", d_smallestKClasses[i]);
	// 	}
	// 	printf("\n");
	// }
}

// TODO call this with the correct dimensions // TODO remove sizeOfSmallest? the block dims make it impossible
__global__ void makePredictions(int *d_predictions, float *d_smallestK, int *d_smallestKClasses, int numInstances, int k) {
	int tid_x = blockIdx.x*blockDim.x + threadIdx.x;
	int tid_y  = blockIdx.y*blockDim.y + threadIdx.y;
	// int startingDistanceIndex = ((tid_y * gridDim.x) + (tid_x/blockDim.x)) * k; // NOTE startingDistanceIndex is used to access the proper "row" of the smallestK matrix
	int startingDistanceIndex = ((tid_y * gridDim.x) * (blockDim.x/k)) * k; // NOTE startingDistanceIndex is used to access the proper "row" of the smallestK matrix
	// TODO test at higher instances values

	if (tid_x == 0 && tid_y == 0) {
		printf("\nin makePredictions blockDim.x: %d, blockDim.y: %d, gridDim.x: %d, gridDim.y: %d:\n\n", blockDim.x, blockDim.y, gridDim.x, gridDim.y);
	}

	int prevS = blockDim.x; // TODO works at max k?
	// if (tid_x == 0 && tid_y == 0) {
	// 	printf("prevS: %d\n", prevS);
	// }

	// if (tid_x == 0 && tid_y == 0) {
	// 	printf("\n first Instance\n");
	// 	for (int i = 0; i < prevS; i++) {
	// 		printf("%f, ", d_smallestK[i]);
	// 	}
	// }

	for (int s = (((blockDim.x + k - 1) / k) / 2) * k; s < prevS; s = (((s / k) + 2 - 1) / 2) * k) { // (ceil(blocksSizeK left / 2) * k)  TODO what happens when k > blockDim?
		if (threadIdx.x < s && threadIdx.x % k == 0 && tid_y < numInstances) {
			int leftIndex = threadIdx.x + startingDistanceIndex;
			int rightIndex = leftIndex + s;

			// if (tid_x == 0 && tid_y == 335) {
			// 	// int endingDistanceIndex = startingDistanceIndex + k;
			// 	// printf("\n d_smallestK, prevS: %d, leftIndex: %d, rightIndex: %d:\n", prevS, leftIndex, rightIndex); // LEFT SHOULD BE 10!
			// 	// for (int i = leftIndex; i < leftIndex + k; i++) {
			// 	// 	printf("%f, ", d_smallestK[i]);
			// 	// }
			// 	printf("\n d_smallestK, prevS: %d, rightIndex: %d:\n", prevS, rightIndex);
			// 	for (int i = rightIndex; i < rightIndex + k; i++) {
			// 		printf("%f, ", d_smallestK[i]);
			// 	}
			// 	// printf("\nhardcoded:\n");
			// 	// printf("d_smallestK[0]: %f\n", d_smallestK[0]);
			// 	// printf("d_smallestK[1]: %f\n", d_smallestK[1]);
			// 	// printf("d_smallestK[2]: %f\n", d_smallestK[2]);
			// 	// printf("d_smallestK[3]: %f\n", d_smallestK[3]);
			// 	// printf("d_smallestK[4]: %f\n", d_smallestK[4]);

			// 	printf("\n\n");

			// 	// printf("\n d_smallestKClasses, prevS: %d:\n", prevS);
			// 	// for (int i = rightIndex; i < rightIndex + k; i++) {
			// 	// 	printf("%d, ", d_smallestKClasses[i]);
			// 	// }
			// 	// printf("\n\n");
			// }

			// printf("s: %d, tid_y: %d,  leftIndex: %d, rightIndex: %d\n", s, tid_y, leftIndex, rightIndex);
			float* result = new float[k]; // TODO does something need to be freed?
			int* resultClasses = new int[k]; // TODO does something need to be freed?

			for (int i = 0; i < k; i++) {
				// if (tid_x == 0 && tid_y == 335)
				// 		printf("\ninside leftIndex: %d, d_smallestK[leftIndex]: %f ...  rightIndex: %d, d_smallestK[rightIndex]: %f\n", leftIndex, d_smallestK[leftIndex], rightIndex, d_smallestK[rightIndex]);

				if (d_smallestK[rightIndex] < d_smallestK[leftIndex]) {
					// if (tid_x == 0 && tid_y == 335)
					// 	printf("\n used right\n");//inside rightIndex: %d, d_smallestK[rightIndex]: %f\n", rightIndex, d_smallestK[rightIndex]);

					result[i] = d_smallestK[rightIndex];
					resultClasses[i] = d_smallestKClasses[rightIndex];
					rightIndex++;
				} else {
					// if (tid_x == 0 && tid_y == 335)
					// 	printf("\n used left\n");//inside leftIndex: %d, d_smallestK[leftIndex]: %f\n", leftIndex, d_smallestK[leftIndex]);

					result[i] = d_smallestK[leftIndex];
					resultClasses[i] = d_smallestKClasses[leftIndex];
					leftIndex++;
				}
				
				// if (tid_x == 0 && tid_y == 1) {
				// 	// int endingDistanceIndex = startingDistanceIndex + k;
				// 	printf("\n after merge d_smallestK, prevS: %d, leftIndex: %d, rightIndex: %d:\n", prevS, leftIndex, rightIndex);
				// 	for (int i = leftIndex; i < leftIndex + k; i++) {
				// 		printf("leftIndex: %d, d_smallestK[leftIndex]: %f\n", leftIndex, d_smallestK[leftIndex]);
				// 	}
				// }
			}

			// int startingKIndex = ((tid_y * gridDim.x) + (tid_x/blockDim.x)) * k;
			int endingKIndex = startingDistanceIndex + k;
			// if (tid_x == 0 && tid_y == 1) {
			// 	printf("\ntid_y: %d, threadIdx.x: %d, startingKIndex: %d\n", tid_y, threadIdx.x, startingDistanceIndex);
			// 	// printf("\nresult:\n");
			// 	// for (int i = 0; i < k; i++) {
			// 	// 	printf("%f, ", result[i]);
			// 	// }
			// 	// printf("\nresultClasses:\n");
			// 	// for (int i = 0; i < k; i++) {
			// 	// 	printf("%d, ", resultClasses[i]);
			// 	// }
			// 	printf("\n");
			// }

			// for (int i = startingDistanceIndex; i < endingKIndex; i++) {
			for (int i = 0; i < k; i++) {
				// printf("tid_y: %d, threadIdx.x + startingDistanceIndex + i: %d, i: %d, result: %d\n", tid_y, threadIdx.x + startingDistanceIndex + i, i, result[i]);
				d_smallestK[threadIdx.x + startingDistanceIndex + i] = result[i];
				d_smallestKClasses[threadIdx.x + startingDistanceIndex + i] = resultClasses[i];
			}
		}
		
		prevS = s;
		__syncthreads();
	}

	// if (tid_x == 0 && tid_y == 335) {
	// 	printf("\n final reduced d_smallestK:\n"); // THESE SHOULD BE THE REDUCED RESULTS NOW
	// 	for (int i = 0; i < 4 * k; i++) {
	// 		printf("%f, ", d_smallestK[i]);
	// 	}
	// 	printf("\n\n");

	// 	printf("\n final reduced d_smallestKClasses:\n");
	// 	for (int i = 0; i < 4 * k; i++) {
	// 		printf("%d, ", d_smallestKClasses[i]);
	// 	}
	// 	printf("\n\n");
	// }

	// __syncthreads(); // TODO delete with above print block

	// make predictions
	// get max class
	if (threadIdx.x == 0 && threadIdx.y < numInstances) {
		int endingDistanceIndex = startingDistanceIndex + k;
		// printf("making prediction, \n");
		int maxClass = 0;
		for (int i = startingDistanceIndex; i < endingDistanceIndex; i++) {  // TODO when 2d make startingDistanceIndex be the start
			if (d_smallestKClasses[i] > maxClass)
				maxClass = d_smallestKClasses[i];
		}
		// printf("maxClass: %d\n", maxClass);

		int* classCounter = new int[maxClass + 1]; // TODO does something need to be freed?
		for (int i = startingDistanceIndex; i < endingDistanceIndex; i++) { // TODO when 2d make startingDistanceIndex be the start
			classCounter[d_smallestKClasses[i]]++;
		}

		// if (tid_x == 0 && tid_y == 0) {
		// 	printf("\n classCounter:\n");
		// 	for (int i = 0; i <= maxClass; i++) {
		// 		printf("%d, ", classCounter[i]);
		// 	}
		// 	printf("\n");
		// }
		
		int voteResult = -1;
		int numberOfVotes = -1;
		for (int i = 0; i <= maxClass; i++) {
			if (classCounter[i] > numberOfVotes) {
				numberOfVotes = classCounter[i];
				voteResult = i;
			}
		d_predictions[tid_y] = voteResult;
		}
		
		// d_predictions[tid_y] = 0;
	}
}

int* computeConfusionMatrix(int* predictions, ArffData* dataset) {
    int* confusionMatrix = (int*)calloc(dataset->num_classes() * dataset->num_classes(), sizeof(int)); // matrix size numberClasses x numberClasses
    
    for(int i = 0; i < dataset->num_instances(); i++) { // for each instance compare the true class and predicted class
        int trueClass = dataset->get_instance(i)->get(dataset->num_attributes() - 1)->operator int32();
        int predictedClass = predictions[i];
        
        confusionMatrix[trueClass*dataset->num_classes() + predictedClass]++;
    }
    
    return confusionMatrix;
}

float computeAccuracy(int* confusionMatrix, ArffData* dataset) {
    int successfulPredictions = 0;
    
    for(int i = 0; i < dataset->num_classes(); i++) {
        successfulPredictions += confusionMatrix[i*dataset->num_classes() + i]; // elements in the diagonal are correct predictions
    }
    
    return successfulPredictions / (float) dataset->num_instances();
}

int main(int argc, char* argv[])
{
	if(argc != 3) {
        printf("Usage: ./main datasets/datasetFile.arff kValue");
        exit(0);
	}
	    
    // Open the dataset
    ArffParser parser(argv[1]);
	ArffData *dataset = parser.parse();
	int k = atoi(argv[2]);
	if (k > dataset->num_instances())
		k = dataset->num_instances();
	
	int datasetMatrixLength = dataset->num_instances();// TODO are tehse needed?
	int datasetMatrixWidth = dataset->num_attributes();
	int numElements = datasetMatrixLength * datasetMatrixWidth;

	// Allocate host memory
	float *h_datasetArray = (float *)malloc(numElements * sizeof(float));
	float *h_distanceMatrix = (float *)malloc(dataset->num_instances() * dataset->num_instances() * sizeof(float)); // used to find all distances in parallel
	int *h_predictions = (int *)malloc(dataset->num_instances() * sizeof(int));

	// Initialize the host input matrixs
	for (int i = 0; i < datasetMatrixLength; ++i) {
		for (int j = 0; j < datasetMatrixWidth; ++j) {
			h_datasetArray[i * datasetMatrixWidth + j] = dataset->get_instance(i)->get(j)->operator float(); // TODO how to handle class?
		}
	}

	// Allocate the device input matrix A
	float *d_datasetArray;
	float *d_distanceMatrix;
	int *d_predictions;

	hipMalloc(&d_datasetArray, numElements * sizeof(float));
	hipMalloc(&d_distanceMatrix, dataset->num_instances() * dataset->num_instances() * sizeof(float));
	hipMalloc(&d_predictions, dataset->num_instances() * sizeof(int));

	// Copy the host input matrixs A and B in host memory to the device input matrixs in
	hipMemcpy(d_datasetArray, h_datasetArray, numElements * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_distanceMatrix, h_distanceMatrix, dataset->num_instances() * dataset->num_instances() * sizeof(float), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float milliseconds = 0;

	// distance matrix
	int gridDimSize = (dataset->num_instances() + DISTANCETHREADSPERBLOCKDIM - 1) / DISTANCETHREADSPERBLOCKDIM;

	dim3 blockSize(DISTANCETHREADSPERBLOCKDIM, DISTANCETHREADSPERBLOCKDIM);
	dim3 gridSize(gridDimSize, gridDimSize);

	printf("CUDA kernel launch with %dx%d blocks of %dx%d threads\n", gridDimSize, gridDimSize, DISTANCETHREADSPERBLOCKDIM, DISTANCETHREADSPERBLOCKDIM);
	
	hipEventRecord(start);

	fillDistanceMatrix<<<gridSize, blockSize>>>(d_datasetArray, d_distanceMatrix, dataset->num_instances(), dataset->num_attributes());

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("GPU time to fill Distance Matrix %f ms\n", milliseconds);



	// matrix reduction
	// reductionBlocksPerGridX is number of blocks (where block has 256 threads) to do all elements in row (dataset->num_instances())
	int reductionBlocksPerGridX = (dataset->num_instances() + REDUCTIONTHREADSBLOCKDIMX - 1) / REDUCTIONTHREADSBLOCKDIMX;
	int reductionBlocksPerGridY = (dataset->num_instances() + REDUCTIONTHREADSBLOCKDIMY - 1) / REDUCTIONTHREADSBLOCKDIMY;

	printf("\nreduction dims: REDUCTIONTHREADSBLOCKDIMX: %d, REDUCTIONTHREADSBLOCKDIMY: %d, reductionBlocksPerGridX: %d, reductionBlocksPerGridY: %d\n",
	REDUCTIONTHREADSBLOCKDIMX, REDUCTIONTHREADSBLOCKDIMY, reductionBlocksPerGridX, reductionBlocksPerGridY);

	dim3 reductionBlockSize(REDUCTIONTHREADSBLOCKDIMX, REDUCTIONTHREADSBLOCKDIMY); // 256 * 4 <= 1024
	dim3 reductionGridSize(reductionBlocksPerGridX, reductionBlocksPerGridY);

	int *h_actualClasses = (int *)malloc(dataset->num_instances() * sizeof(int));
	for (int i = 0; i < dataset->num_instances(); i++) {
		h_actualClasses[i] = dataset->get_instance(i)->get(dataset->num_attributes() - 1)->operator int32();
	}

	int *d_actualClasses;
	hipMalloc(&d_actualClasses, dataset->num_instances() * sizeof(int));

	hipMemcpy(d_actualClasses, h_actualClasses, dataset->num_instances() * sizeof(int), hipMemcpyHostToDevice);

	// float **d_smallestK;
	float *d_smallestK;
	hipMalloc(&d_smallestK, k * reductionBlocksPerGridX * dataset->num_instances() * sizeof(float));
	int *d_smallestKClasses;
	hipMalloc(&d_smallestKClasses, k * reductionBlocksPerGridX * dataset->num_instances() * sizeof(int));

	// cudadevicesynchronize(); // wait for distanceMAtrix to be filled TODO needed?

	deviceFindMinK<<<reductionGridSize, reductionBlockSize>>>(d_smallestK, d_smallestKClasses, d_distanceMatrix, d_actualClasses, dataset->num_instances(), k);

	hipError_t hipError_t = hipGetLastError();
	if(hipError_t != hipSuccess) {
		fprintf(stderr, "post deviceFindMinK hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
		exit(EXIT_FAILURE);
	}

	// cudadevicesynchronize(); // wait for smallestK's to be filled TODO needed?



	// prediction matrix
	int numberOfKSegmentsPerRow = k * reductionBlocksPerGridX;
	int maxNumberOFInstancesToPredictAtOnce = 1024 / numberOfKSegmentsPerRow;

	int predictionBlocksPerGridX = (numberOfKSegmentsPerRow + 1024 - 1) / 1024;
	int predictionBlocksPerGridY = (dataset->num_instances() + maxNumberOFInstancesToPredictAtOnce - 1) / maxNumberOFInstancesToPredictAtOnce;

	printf("\nprediction dims: numberOfKSegmentsPerRow: %d, maxNumberOFInstancesToPredictAtOnce: %d, predictionBlocksPerGridX: %d, predictionBlocksPerGridY: %d\n",
		numberOfKSegmentsPerRow, maxNumberOFInstancesToPredictAtOnce, predictionBlocksPerGridX, predictionBlocksPerGridY);

	dim3 predictionBlockSize(numberOfKSegmentsPerRow, maxNumberOFInstancesToPredictAtOnce);
	dim3 predictionGridSize(predictionBlocksPerGridX, predictionBlocksPerGridY);


	makePredictions<<<predictionGridSize, predictionBlockSize>>>(d_predictions, d_smallestK, d_smallestKClasses, dataset->num_instances(), k);

	hipError_t = hipGetLastError();
	if(hipError_t != hipSuccess) {
		fprintf(stderr, "post makePredictions hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
		exit(EXIT_FAILURE);
	}

	hipMemcpy(h_predictions, d_predictions, dataset->num_instances() * sizeof(int), hipMemcpyDeviceToHost);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("time to make predictions %f ms\n", milliseconds);






	// for (int i = 0; i < dataset->num_instances(); i++) {
	// 	printf("actual: %d, predicted: %d\n", h_actualClasses[i], h_predictions[i]);
	// }

	// Compute the confusion matrix
	int* confusionMatrix = computeConfusionMatrix(h_predictions, dataset);
	// Calculate the accuracy
	float accuracy = computeAccuracy(confusionMatrix, dataset);

	printf("The KNN classifier for %lu instances with k=%d had an accuracy of %.4f\n", dataset->num_instances(), k, accuracy);

	// Free device global memory
	hipFree(d_datasetArray);
	hipFree(d_predictions);
	hipFree(d_smallestK);
	hipFree(d_smallestKClasses);
	hipFree(d_actualClasses);

	// Free host memory
	free(h_datasetArray);
	free(h_predictions);
	free(h_actualClasses);

	return 0;
}