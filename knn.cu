#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <map>
#include <climits> 
#include <cfloat>
#include <vector>
#include <algorithm> // for heap
#include <numeric> // std::iota
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include "libarff/arff_parser.h"
#include "libarff/arff_data.h"

#define THREADSPERBLOCK 256
#define YINDEXTOCHECK 335

__global__ void fillDistanceMatrix(float *d_datasetArray, float *d_distanceMatrix, int width, int numberOfAttributes) { // d_distanceMatrix is a square matrix
	int column = ( blockDim.x * blockIdx.x ) + threadIdx.x;
	int row    = ( blockDim.y * blockIdx.y ) + threadIdx.y;

	if (row < width && column < width) {
		if (row == column) {
			d_distanceMatrix[row*width + column] = FLT_MAX; // cant compare to to self
		} else {
			float distance = 0;

			for(int k = 0; k < numberOfAttributes - 1; k++) { // compute the distance between the two instances
				float diff = d_datasetArray[row * numberOfAttributes + k] - d_datasetArray[column * numberOfAttributes + k]; // one instance minus the other
				distance += diff * diff;
			}

			d_distanceMatrix[row*width + column] = sqrt(distance);
		}
	}
}

// Uses a shared memory reduction approach to find the smallest k values
__global__ void deviceFindMinK(float *d_smallestK, int *d_smallestKClasses, float *d_distanceMatrix, int startingDistanceIndex, int yIndex, int *d_actualClasses, int numInstances, int k) {
	__shared__ float sharedDistanceMemory[THREADSPERBLOCK];
	__shared__ int sharedClassMemory[THREADSPERBLOCK];

	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	if (tid == 0 && yIndex == 0) {
		printf("\nin deviceFindMinK blockDim.x: %d, gridDim.x: %d:\n", blockDim.x, gridDim.x);
	}

	sharedDistanceMemory[threadIdx.x] = (tid < numInstances) ? d_distanceMatrix[startingDistanceIndex + tid] : FLT_MAX; // NOTE startingDistanceIndex is used to accesss the proper "row" of the matrix
	sharedClassMemory[threadIdx.x] = (tid < numInstances) ? d_actualClasses[tid] : -1;
	
	__syncthreads();
	
	if (threadIdx.x == 0 && yIndex == YINDEXTOCHECK) {
		printf("\n pre shared mem. k = %d, blockIdx.x: %d:\n", k, blockIdx.x);
		for (int i = 0; i < THREADSPERBLOCK; i++) {
			printf("%f, ", sharedDistanceMemory[i]);
		}
		
		printf("\n pre shared class mem. k = %d, blockIdx.x: %d:\n", k, blockIdx.x);
		for (int i = 0; i < THREADSPERBLOCK; i++) {
			printf("%d, ", sharedClassMemory[i]);
		}
		printf("\n\n");
	}
    __syncthreads();

	// do reduction in shared memory
	// for (int s = 0; s < blockDim.x; s += k) {
	// ((ceiling division of blockDim.x / k) / 2) is number of "chunks" of size k that barely spill over the halfway point
	// mulitply it by k to get the actual max s value to start at
	int prevS = blockDim.x; // TODO works at max k?

	for (int s = (((blockDim.x + k - 1) / k) / 2) * k; s < prevS; s = (((s / k) + 2 - 1) / 2) * k) { // (ceil(blocksSizeK left / 2) * k)  TODO what happens when k > blockDim?
		if (threadIdx.x < s && threadIdx.x % k == 0) {
			int leftIndex = threadIdx.x;
			int rightIndex = leftIndex + s;
			float* result = new float[k]; // TODO does something need to be freed?
			int* resultClasses = new int[k]; // TODO does something need to be freed?

			// if on first iteration
			if (prevS == blockDim.x) {
				thrust::sort_by_key(thrust::seq, sharedDistanceMemory + leftIndex, sharedDistanceMemory + leftIndex + k, sharedClassMemory + leftIndex);
				int actualEndingIndex = rightIndex + k;
				if (actualEndingIndex >= THREADSPERBLOCK)
					actualEndingIndex = THREADSPERBLOCK;
				thrust::sort_by_key(thrust::seq, sharedDistanceMemory + rightIndex, sharedDistanceMemory + actualEndingIndex, sharedClassMemory + rightIndex);
			}

			for (int i = 0; i < k; i++) {
				if (rightIndex < blockDim.x && sharedDistanceMemory[rightIndex] < sharedDistanceMemory[leftIndex]) {
					result[i] = sharedDistanceMemory[rightIndex];
					resultClasses[i] = sharedClassMemory[rightIndex];
					rightIndex++;
				} else {
					result[i] = sharedDistanceMemory[leftIndex];
					resultClasses[i] = sharedClassMemory[leftIndex];
					leftIndex++;
				}
			}

			for (int i = 0; i < k; i++) {
				sharedDistanceMemory[threadIdx.x + i] = result[i];
				sharedClassMemory[threadIdx.x + i] = resultClasses[i];
			}
		}
		
		prevS = s;

		__syncthreads();
	}

	if (threadIdx.x == 0 && blockIdx.x == gridDim.x - 1 && yIndex == YINDEXTOCHECK) {
		printf("\n block final smallest k:\n");
		for (int i = 0; i < k; i++) {
			printf("%f, ", sharedDistanceMemory[i]);
		}
		printf("\n\n");
		printf("\n block final classes of smallest k:\n");
		for (int i = 0; i < k; i++) {
			printf("%d, ", sharedClassMemory[i]);
		}
		printf("\n");
	}

	// write your nearestK to global mem
	if (threadIdx.x == 0) {
		// int startingKIndex = k  * yIndex;
		int startingKIndex = ((yIndex * gridDim.x) + (tid/blockDim.x)) * k;
		int endingKIndex = startingKIndex + k;
	
		if (threadIdx.x == 0 && blockIdx.x == gridDim.x - 1 && yIndex == YINDEXTOCHECK)
			printf("writing final block to global startingKIndex: %d, endingKIndex: %d\n", startingKIndex, endingKIndex);
	
			int j = 0;
		for (int i = startingKIndex; i < endingKIndex; i++) {
			d_smallestK[i] = sharedDistanceMemory[j];
			d_smallestKClasses[i] = sharedClassMemory[j];
			j++;
		}
	}

	if (threadIdx.x == 0 && blockIdx.x == gridDim.x - 1 && yIndex == YINDEXTOCHECK) {
		// int startingKIndex = k * yIndex;
		// in deviceFindMinK blockDim.x: 256, blockDim.y: 4, gridDim.x: 2, gridDim.y: 84:
		int startingKIndex = ((yIndex * gridDim.x) + (tid/blockDim.x)) * k;
		int endingKIndex = startingKIndex + k;
		printf("\n distance final iteration startingKIndex: %d, endingKIndex: %d\n", startingKIndex, endingKIndex);
		startingKIndex = startingKIndex - k * (gridDim.x - blockIdx.x);
		printf(" shown distance final iteration startingKIndex: %d, endingKIndex: %d\n", startingKIndex, endingKIndex);
		printf("distances:\n");
		for (int i = startingKIndex; i < endingKIndex; i++) {
			printf("%f, ", d_smallestK[i]);
		}
		printf("\n\n");

		printf("classes:\n");
		for (int i = startingKIndex; i < endingKIndex; i++) {
			printf("%d, ", d_smallestKClasses[i]);
		}
		printf("\n\n");
	}
}

// TODO call this with the correct dimensions
__global__ void makePredictions(int *d_predictions, float *d_smallestK, int *d_smallestKClasses, int sizeOfSmallest, int numberOfKSegmentsPerRow, int yIndex, int k) {
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	int startingDistanceIndex = (numberOfKSegmentsPerRow) * yIndex;

	if (tid == 0 && yIndex == 0) {
		printf("\nin makePredictions blockDim.x: %d, gridDim.x: %d:\n", blockDim.x, gridDim.x);
	}

	if (tid == 0 && yIndex == YINDEXTOCHECK) {
		printf("final startingDistanceIndex:%d\n", startingDistanceIndex);
	}

	if (threadIdx.x == 0 && blockIdx.x == gridDim.x - 1 && yIndex == YINDEXTOCHECK) {
		int startingKIndex = startingDistanceIndex; // ((yIndex * gridDim.x) + (tid/blockDim.x)) * k;
		int endingKIndex = startingKIndex + (blockDim.x * gridDim.x); // + k;
		printf(" prediction pre reduction iteration startingKIndex: %d, endingKIndex: %d\n", startingKIndex, endingKIndex);
		printf("distances:\n");
		for (int i = startingKIndex; i < endingKIndex; i++) {
			printf("%f, ", d_smallestK[i]);
		}
		printf("\n\n");

		printf("classes:\n");
		for (int i = startingKIndex; i < endingKIndex; i++) {
			printf("%d, ", d_smallestKClasses[i]);
		}
		printf("\n\n");
	}

	int prevS = blockDim.x;

	for (int s = (((prevS + k - 1) / k) / 2) * k; s < prevS; s = (((s / k) + 2 - 1) / 2) * k) { // (ceil(blocksSizeK left / 2) * k)  TODO what happens when k > blockDim?
		if (threadIdx.x < s && threadIdx.x % k == 0 && threadIdx.x < sizeOfSmallest) {
			int leftIndex = threadIdx.x + startingDistanceIndex;
			int rightIndex = leftIndex + s;
			if (threadIdx.x == 0 && yIndex == YINDEXTOCHECK) {
				printf("prediction tid: %d, prevS: %d, s:%d, leftIndex: %d, rightIndex: %d\n", tid, prevS, s, leftIndex, rightIndex);
			}
			float* result = new float[k]; // TODO does something need to be freed?
			int* resultClasses = new int[k]; // TODO does something need to be freed?

			for (int i = 0; i < k; i++) {
				if (threadIdx.x == 0 && yIndex == YINDEXTOCHECK)
					printf("tid: %d, prevS: %d, s:%d, leftIndex: %d, leftdistance: %f, rightIndex: %d, rightdistance: %f\n", tid, prevS, s, leftIndex, d_smallestK[leftIndex], rightIndex, d_smallestK[rightIndex]);
				if (rightIndex < (startingDistanceIndex + (blockDim.x * gridDim.x)) && d_smallestK[rightIndex] < d_smallestK[leftIndex]) {
					result[i] = d_smallestK[rightIndex];
					resultClasses[i] = d_smallestKClasses[rightIndex];
					rightIndex++;

					if (threadIdx.x == 0 && yIndex == YINDEXTOCHECK)
						printf("used rightIndex i: %d, result[i]: %f\n", i, result[i]);//: rightIndex: %d, distance: %f\n", rightIndex, d_smallestK[rightIndex]);
				} else {
					result[i] = d_smallestK[leftIndex];
					resultClasses[i] = d_smallestKClasses[leftIndex];
					leftIndex++;

					if (threadIdx.x == 0 && yIndex == YINDEXTOCHECK)
						printf("used leftIndex i: %d, result[i]: %f\n", i, result[i]);//: leftIndex: %d, distance: %f\n", leftIndex, d_smallestK[leftIndex]);
				}
			}

			for (int i = 0; i < k; i++) {
				d_smallestK[threadIdx.x + startingDistanceIndex + i] = result[i];
				d_smallestKClasses[threadIdx.x + startingDistanceIndex + i] = resultClasses[i];
			}
		}
		
		prevS = s;
		__syncthreads();
	}

	if (threadIdx.x == 0 && blockIdx.x == gridDim.x - 1 && yIndex == YINDEXTOCHECK) {
		// int startingKIndex = k * yIndex;
		// in deviceFindMinK blockDim.x: 256, blockDim.y: 4, gridDim.x: 2, gridDim.y: 84:
		int startingKIndex = startingDistanceIndex; // ((yIndex * gridDim.x) + (tid/blockDim.x)) * k;
		int endingKIndex = startingKIndex + (blockDim.x * gridDim.x); // + k;
		printf(" prediction final iteration startingKIndex: %d, endingKIndex: %d\n", startingKIndex, endingKIndex);
		endingKIndex = startingKIndex + k;
		printf(" shown prediction final iteration endingKIndex: %d, endingKIndex: %d\n", startingKIndex, endingKIndex);
		printf("distances:\n");
		for (int i = startingKIndex; i < endingKIndex; i++) {
			printf("%f, ", d_smallestK[i]);
		}
		printf("\n\n");

		printf("classes:\n");
		for (int i = startingKIndex; i < endingKIndex; i++) {
			printf("%d, ", d_smallestKClasses[i]);
		}
		printf("\n\n");
	}

	// make predictions
	// get max class
	if (threadIdx.x == 0) {
		int endingDistanceIndex = startingDistanceIndex + k;
		// printf("making prediction, \n");
		int maxClass = 0;
		for (int i = startingDistanceIndex; i < endingDistanceIndex; i++) {  // TODO when 2d make startingDistanceIndex be the start
			if (d_smallestKClasses[i] > maxClass)
				maxClass = d_smallestKClasses[i];
		}
		// printf("maxClass: %d\n", maxClass);

		int* classCounter = new int[maxClass + 1]; // TODO does something need to be freed?
		for (int i = startingDistanceIndex; i < endingDistanceIndex; i++) { // TODO when 2d make startingDistanceIndex be the start
			classCounter[d_smallestKClasses[i]]++;
		}

		// printf("\n classCounter:\n");
		// for (int i = 0; i <= maxClass; i++) {
		// 	printf("%d, ", classCounter[i]);
		// }
		// printf("\n\n");
		
		int voteResult = -1;
		int numberOfVotes = -1;
		for (int i = 0; i <= maxClass; i++) {
			if (classCounter[i] > numberOfVotes) {
				numberOfVotes = classCounter[i];
				voteResult = i;
			}
		}
		
		d_predictions[yIndex] = voteResult;
	}
}

int* computeConfusionMatrix(int* predictions, ArffData* dataset) {
    int* confusionMatrix = (int*)calloc(dataset->num_classes() * dataset->num_classes(), sizeof(int)); // matrix size numberClasses x numberClasses
    
    for(int i = 0; i < dataset->num_instances(); i++) { // for each instance compare the true class and predicted class
        int trueClass = dataset->get_instance(i)->get(dataset->num_attributes() - 1)->operator int32();
        int predictedClass = predictions[i];
        
        confusionMatrix[trueClass*dataset->num_classes() + predictedClass]++;
    }
    
    return confusionMatrix;
}

float computeAccuracy(int* confusionMatrix, ArffData* dataset) {
    int successfulPredictions = 0;
    
    for(int i = 0; i < dataset->num_classes(); i++) {
        successfulPredictions += confusionMatrix[i*dataset->num_classes() + i]; // elements in the diagonal are correct predictions
    }
    
    return successfulPredictions / (float) dataset->num_instances();
}

int main(int argc, char* argv[])
{
	if(argc != 3) {
        printf("Usage: ./main datasets/datasetFile.arff kValue");
        exit(0);
	}
	    
    // Open the dataset
    ArffParser parser(argv[1]);
	ArffData *dataset = parser.parse();
	int k = atoi(argv[2]);
	if (k > dataset->num_instances())
		k = dataset->num_instances();
	
	int datasetMatrixLength = dataset->num_instances();// TODO are tehse needed?
	int datasetMatrixWidth = dataset->num_attributes();
	int numElements = datasetMatrixLength * datasetMatrixWidth;

	// Allocate host memory
	float *h_datasetArray = (float *)malloc(numElements * sizeof(float));
	float *h_distanceMatrix = (float *)malloc(dataset->num_instances() * dataset->num_instances() * sizeof(float)); // used to find all distances in parallel
	int *h_predictions = (int *)malloc(dataset->num_instances() * sizeof(int));
	int *h_predictions_CPUres = (int *)malloc(dataset->num_instances() * sizeof(int)); // TODO do I need a cpu version?

	// Initialize the host input matrixs
	for (int i = 0; i < datasetMatrixLength; ++i) {
		for (int j = 0; j < datasetMatrixWidth; ++j) {
			h_datasetArray[i * datasetMatrixWidth + j] = dataset->get_instance(i)->get(j)->operator float(); // TODO how to handle class?
		}
	}

	// Allocate the device input matrix A
	float *d_datasetArray;
	float *d_distanceMatrix;
	int *d_predictions;

	hipMalloc(&d_datasetArray, numElements * sizeof(float));
	hipMalloc(&d_distanceMatrix, dataset->num_instances() * dataset->num_instances() * sizeof(float));
	hipMalloc(&d_predictions, dataset->num_instances() * sizeof(int));

	// Copy the host input matrixs A and B in host memory to the device input matrixs in
	hipMemcpy(d_datasetArray, h_datasetArray, numElements * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_distanceMatrix, h_distanceMatrix, dataset->num_instances() * dataset->num_instances() * sizeof(float), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float milliseconds = 0;

	int threadsPerBlockDim = 16; // dataset->num_attributes(); // so each thread handles 1 attribute. TODO handle class
	int gridDimSize = (dataset->num_instances() + threadsPerBlockDim - 1) / threadsPerBlockDim; //512 / threadsPerBlockDim; // TODO try other values?  // TODO no clue. maybe match num attributes? was matrixSize

	// this is all for the distance matrix
	dim3 blockSize(threadsPerBlockDim, threadsPerBlockDim);
	dim3 gridSize(gridDimSize, gridDimSize);

	printf("CUDA kernel launch with %dx%d blocks of %dx%d threads\n", gridDimSize, gridDimSize, threadsPerBlockDim, threadsPerBlockDim);
	
	hipEventRecord(start);

	fillDistanceMatrix<<<gridSize, blockSize>>>(d_datasetArray, d_distanceMatrix, dataset->num_instances(), dataset->num_attributes());

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("GPU time to fill Distance Matrix %f ms\n", milliseconds);



	// this is all for the matrix reduction TODO delete?
	int blocksPerGrid = (dataset->num_instances() + THREADSPERBLOCK - 1) / THREADSPERBLOCK;
	printf("THREADSPERBLOCK: %d, blocksPerGrid: %d\n", THREADSPERBLOCK, blocksPerGrid);

	int *h_actualClasses = (int *)malloc(dataset->num_instances() * sizeof(int));
	for (int i = 0; i < dataset->num_instances(); i++) {
		h_actualClasses[i] = dataset->get_instance(i)->get(dataset->num_attributes() - 1)->operator int32();
	}

	int *d_actualClasses;
	hipMalloc(&d_actualClasses, dataset->num_instances() * sizeof(int));

	hipMemcpy(d_actualClasses, h_actualClasses, dataset->num_instances() * sizeof(int), hipMemcpyHostToDevice);

	float *d_smallestK;
	hipMalloc(&d_smallestK, k * blocksPerGrid * dataset->num_instances() * sizeof(float));
	int *d_smallestKClasses;
	hipMalloc(&d_smallestKClasses, k * blocksPerGrid * dataset->num_instances() * sizeof(int));

	printf("smallestSize: %d\n", k * blocksPerGrid * dataset->num_instances());

	// cudadevicesynchronize();

	for (int i = 0; i < dataset->num_instances(); i++) {
		deviceFindMinK<<<blocksPerGrid, THREADSPERBLOCK>>>(d_smallestK, d_smallestKClasses, d_distanceMatrix, i * dataset->num_instances(), i, d_actualClasses, dataset->num_instances(), k);

		hipError_t hipError_t = hipGetLastError();
		if(hipError_t != hipSuccess) {
			fprintf(stderr, "post deviceFindMinK hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
			exit(EXIT_FAILURE);
		}
	}

	// cudadevicesynchronize(); // wait for smallestK's to be filled TODO needed?
	int predictionThreadsPerBlock = k * blocksPerGrid;
	int predictionBlocksPerGrid = (blocksPerGrid + 1024 - 1) / 1024;
	int numberOfKSegmentsPerRow = k * blocksPerGrid;

	int sizeOfSmallest = k * blocksPerGrid * dataset->num_instances();
	for (int i = 0; i < dataset->num_instances(); i++) {
		// TODO change dimensions
		makePredictions<<<predictionBlocksPerGrid, predictionThreadsPerBlock>>>(d_predictions, d_smallestK, d_smallestKClasses, sizeOfSmallest, numberOfKSegmentsPerRow, i, k);

		hipError_t hipError_t = hipGetLastError();
		if(hipError_t != hipSuccess) {
			fprintf(stderr, "post makePredictions hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
			exit(EXIT_FAILURE);
		}
	}

	hipMemcpy(h_predictions, d_predictions, dataset->num_instances() * sizeof(int), hipMemcpyDeviceToHost);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("\ntime to make predictions %f ms\n", milliseconds);


	// for (int i = 0; i < dataset->num_instances(); i++) {
	// 	printf("actual: %d, predicted: %d\n", h_actualClasses[i], h_predictions[i]);
	// }

	// Compute the confusion matrix
	int* confusionMatrix = computeConfusionMatrix(h_predictions, dataset);
	// Calculate the accuracy
	float accuracy = computeAccuracy(confusionMatrix, dataset);

	printf("The KNN classifier for %lu instances with k=%d had an accuracy of %.4f\n", dataset->num_instances(), k, accuracy);

	// Free device global memory
	hipFree(d_datasetArray);
	hipFree(d_predictions);
	hipFree(d_smallestK);
	hipFree(d_smallestKClasses);
	hipFree(d_actualClasses);

	// Free host memory
	free(h_datasetArray);
	free(h_predictions);
	free(h_predictions_CPUres);
	free(h_actualClasses);

	return 0;
}